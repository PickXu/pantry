#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "common.h"

__global__ void vecadd_kernel(int *C, int *A, int *B, size_t n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        C[i] = A[i] + B[i];
}

void vecadd(int *C, const int *A, const int *B, size_t n)
{
    if (n == 0)
        return;

    int size = sizeof(int) * n;
    int *d_A, *d_B;
    checkCudaErrors(hipMalloc(&d_A, size));
    checkCudaErrors(hipMalloc(&d_B, size));

    checkCudaErrors(hipMemcpy(d_A, A, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_B, B, size, hipMemcpyHostToDevice));

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;

    vecadd_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_A, d_B, n);

    checkCudaErrors(hipMemcpy(C, d_A, size, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_A));
    checkCudaErrors(hipFree(d_B));
}

